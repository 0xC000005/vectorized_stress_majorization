#include "hip/hip_runtime.h"
#include "cuSolver.cuh"
#include "stdafx.h"

#ifndef VIENNACL_WITH_CUDA
#define VIENNACL_WITH_CUDA
#endif

#ifdef min
#undef min
#endif  
#ifdef __INTELLISENSE__
#define __HIPCC__
#endif

#include <iostream>
#include <vector>
#include "viennacl/linalg/cg.hpp"
#include "viennacl/vector.hpp"
#include "viennacl/matrix.hpp"
#include "viennacl/matrix_proxy.hpp"
#include "viennacl/compressed_matrix.hpp"
#include "viennacl/linalg/prod.hpp"

#ifdef __INTELLISENSE__
#undef __global__
#undef __device__
#undef __constant__
#undef __forceinline__
#undef __shared__
#undef __restrict__

#define __global__
#define __device__
#define __constant__
#define __forceinline__
#define __shared__
#define __restrict__
#endif

#define dist(i ,j) ((i)<(j)?dist[((j) - ((i)+1) + ((i)*(2*n-(i)-1))/2)]:dist[((i) - ((j)+1) + ((j)*(2*n-(j)-1))/2)])


/*Rotate and move for Symmetry Constraints*/
__inline__ __device__ void _rotate(float alpha, float &edge_x, float &edge_y);
__inline__ __device__ void moveCenterToZero(float center_x,
	float &edge_x, float &edge_y);
/*Shape matching
* Rotate and transformate the relative_shape
* to map the corresponding nodes in P_Opt*/
__inline__ __device__ void ShapeMatching(float *relative_shape,
	float *P_Opt_x, float *P_Opt_y, int *pid, int node_num);

/*For Equal Angle Constraints*/
__forceinline__ __device__ int EqualAngleKernel(
	void* __restrict__ constraints,
	float* __restrict__ P_Opt_x,
	float* __restrict__ P_Opt_y,
	float* __restrict__ _right_hand_x,
	float* __restrict__ _right_hand_y) {

	int*   i_constraints = (int  *)constraints;
	float* f_constraints = (float *)constraints;
	int pid_size = i_constraints[1], pid_begin = 5;
	float p_length = f_constraints[3];
	float param = f_constraints[4];
	int relative_size = 2 * pid_size, relative_begin = pid_begin + pid_size;

	int warpid = threadIdx.x >> 5;
	int laneid = threadIdx.x - warpid << 5;
	__shared__ int voting[4];
	if (laneid == 0)
		voting[warpid] = i_constraints[2];
	__syncthreads();
	if (voting[warpid] != i_constraints[2])
		voting[warpid] = -1;
	__syncthreads();

	ShapeMatching(&f_constraints[relative_begin], P_Opt_x, P_Opt_y, &i_constraints[pid_begin], pid_size);

	for (int i = 0; i < pid_size - 1; i++) {
		float dx = param * (f_constraints[relative_begin + relative_size - 2] - f_constraints[relative_begin + 2 * i + 0]);
		float dy = param * (f_constraints[relative_begin + relative_size - 1] - f_constraints[relative_begin + 2 * i + 1]);

		float sum = dx, sum2 = dy;
		if (voting[warpid] > 0)
		{
			for (int i = 16; i >= 1; i >>= 1)
			{
				sum += __shfl_down(sum, i);
				sum2 += __shfl_down(sum2, i);
			}
			if (laneid == 0)
			{
				//child
				atomicAdd(&_right_hand_x[i_constraints[pid_begin + i]], -sum);
				atomicAdd(&_right_hand_y[i_constraints[pid_begin + i]], -sum2);
			}
		}
		else {
			//child
			atomicAdd(&_right_hand_x[i_constraints[pid_begin + i]], -dx);
			atomicAdd(&_right_hand_y[i_constraints[pid_begin + i]], -dy);
		}
		//parent
		atomicAdd(&_right_hand_x[i_constraints[pid_begin + pid_size - 1]], dx);
		atomicAdd(&_right_hand_y[i_constraints[pid_begin + pid_size - 1]], dy);
	}

}
/*For Circle Constraints*/
__forceinline__ __device__ int CircleKernel(
	void* __restrict__ constraints,
	float* __restrict__ P_Opt_x,
	float* __restrict__ P_Opt_y,
	float* __restrict__ _right_hand_x,
	float* __restrict__ _right_hand_y) {

	int*   i_constraints = (int  *)constraints;
	float* f_constraints = (float *)constraints;
	float differ_angle = f_constraints[0];
	int pid_size = i_constraints[1], pid_begin = 4;
	int r_lengths_size = i_constraints[2], r_lengths_begin = pid_begin + pid_size;
	int relative_size = 2 * pid_size, relative_begin = r_lengths_begin + r_lengths_size;
	float param = f_constraints[3];

	int warpid = threadIdx.x >> 5;
	int laneid = threadIdx.x - warpid << 5;
	__shared__ int voting[4];
	if (laneid == 0)
		voting[warpid] = i_constraints[2];
	__syncthreads();
	if (voting[warpid] != i_constraints[2])
		voting[warpid] = -1;
	__syncthreads();

	ShapeMatching(&f_constraints[relative_begin], P_Opt_x, P_Opt_y, &i_constraints[pid_begin], pid_size);

	for (int i = 0; i < pid_size; i++) {

		float dx = f_constraints[relative_begin + 2 * ((i + 1) % pid_size) + 0] - f_constraints[relative_begin + 2 * i + 0];
		float dy = f_constraints[relative_begin + 2 * ((i + 1) % pid_size) + 1] - f_constraints[relative_begin + 2 * i + 1];

		//printf("in loop with %d nodes, d=(%f,%f),i=%d,\n ", pid_size, dx, dy, i);

		dx = dx*param / f_constraints[r_lengths_begin + i];
		dy = dy*param / f_constraints[r_lengths_begin + i];

		float sum = dx, sum2 = dy;
		if (voting[warpid] > 0)
		{
			for (int i = 16; i >= 1; i >>= 1)
			{
				sum += __shfl_down(sum, i);
				sum2 += __shfl_down(sum2, i);
			}
			if (laneid == 0)
			{
				//child
				atomicAdd(&_right_hand_x[i_constraints[pid_begin + i]], -sum);
				atomicAdd(&_right_hand_y[i_constraints[pid_begin + i]], -sum2);
			}
		}
		else {
			//child
			atomicAdd(&_right_hand_x[i_constraints[pid_begin + i]], -dx);
			atomicAdd(&_right_hand_y[i_constraints[pid_begin + i]], -dy);
		}
		//parent
		atomicAdd(&_right_hand_x[i_constraints[pid_begin + (i + 1) % pid_size]], dx);
		atomicAdd(&_right_hand_y[i_constraints[pid_begin + (i + 1) % pid_size]], dy);

	}
}
/*For Edge Crossing Remove Constraints*/
__forceinline__ __device__ int CrossingRemovalKernel(
	void* __restrict__ constraints,
	float* __restrict__ P_Opt_x,
	float* __restrict__ P_Opt_y,
	float* __restrict__ _right_hand_x,
	float* __restrict__ _right_hand_y) {

	float *f_constraints = (float *)constraints;
	int*   i_constraints = (int  *)constraints;

	int edge_size = i_constraints[0], edge_begin = 2;
	int rest_length_size = i_constraints[0], rest_length_begin = edge_begin + 2 * edge_size;
	int edge_weights_begin = rest_length_begin + rest_length_size;
	float param = f_constraints[1];

	/*the idea direction is the sum direction of intersecting edges*/
	float  dire_x = 0, dire_y = 0;
	for (int i = 0; i < edge_size; i++) {
		float edge_x = P_Opt_x[i_constraints[edge_begin + i * 2 + 0]] - P_Opt_x[i_constraints[edge_begin + i * 2 + 1]];
		float edge_y = P_Opt_y[i_constraints[edge_begin + i * 2 + 0]] - P_Opt_y[i_constraints[edge_begin + i * 2 + 1]];
		/*keep the relative direction roughly*/
		if (dire_x*edge_x + dire_y*edge_y < 0) {
			dire_x += -edge_x;
			dire_y += -edge_y;
		}
		else {
			dire_x += edge_x;
			dire_y += edge_y;
		}
	}

	float norm = sqrt(dire_x * dire_x + dire_y * dire_y);
	if (isnan(norm)) norm = 1;
	dire_x = dire_x * edge_size * param / norm;
	dire_y = dire_y * edge_size * param / norm;
	for (int i = 0; i < edge_size; i++) {
		float edge_x = P_Opt_x[i_constraints[edge_begin + i * 2 + 0]] - P_Opt_x[i_constraints[edge_begin + i * 2 + 1]];
		float edge_y = P_Opt_y[i_constraints[edge_begin + i * 2 + 0]] - P_Opt_y[i_constraints[edge_begin + i * 2 + 1]];
		float dx = dire_x * f_constraints[edge_weights_begin + i] / f_constraints[rest_length_begin + i];
		float dy = dire_y * f_constraints[edge_weights_begin + i] / f_constraints[rest_length_begin + i];
		/*keep the relative direction roughly*/
		if (dire_x*edge_x + dire_y*edge_y < 0) {
			//source
			atomicAdd(&_right_hand_x[i_constraints[edge_begin + i * 2 + 0]], -dx);
			atomicAdd(&_right_hand_y[i_constraints[edge_begin + i * 2 + 0]], -dy);
			//target
			atomicAdd(&_right_hand_x[i_constraints[edge_begin + i * 2 + 1]], dx);
			atomicAdd(&_right_hand_y[i_constraints[edge_begin + i * 2 + 1]], dy);
		}
		else {
			//source
			atomicAdd(&_right_hand_x[i_constraints[edge_begin + i * 2 + 0]], dx);
			atomicAdd(&_right_hand_y[i_constraints[edge_begin + i * 2 + 0]], dy);
			//target
			atomicAdd(&_right_hand_x[i_constraints[edge_begin + i * 2 + 1]], -dx);
			atomicAdd(&_right_hand_y[i_constraints[edge_begin + i * 2 + 1]], -dy);
		}
	}
	return edge_size * 4 + 2;

}
/*For Node Noverlap Constraints*/
__forceinline__ __device__ int NoverlapKernel(
	void* __restrict__ constraints,
	float* __restrict__ P_Opt_x,
	float* __restrict__ P_Opt_y,
	float* __restrict__ _right_hand_x,
	float* __restrict__ _right_hand_y) {
	float *f_constraints = (float *)constraints;
	int   *i_constraints = (int   *)constraints;

	float x_dis = P_Opt_x[i_constraints[2]] - P_Opt_x[i_constraints[3]];
	float y_dis = P_Opt_y[i_constraints[2]] - P_Opt_y[i_constraints[3]];
	float width = f_constraints[0];
	float height = f_constraints[1];
	int source_node = i_constraints[2], target_node = i_constraints[3];

	float eucli_dis = sqrt(x_dis*x_dis + y_dis*y_dis);
	if (isnan(eucli_dis))
	{
		eucli_dis = 1;
	}
	float param = f_constraints[4] / eucli_dis;
	/*these two nodes are overlap*/
	if (abs(x_dis) < width && abs(y_dis) < height) {
		float dx = 0, dy = 0;
		/*move along x-axis costing less, more desiring move along y-axis*/
		if (2 * (width - abs(x_dis)) < height - abs(y_dis)) {
			dx = f_constraints[4] * width;
			dy = param * abs(y_dis);
			/*keep the relative direction roughly*/
			if (dx*x_dis + dy*y_dis < 0) {
				dx *= -1;
				dy *= -1;
			}
			atomicAdd(&_right_hand_x[source_node], -dx);
			atomicAdd(&_right_hand_y[source_node], -dy);
			atomicAdd(&_right_hand_x[target_node], dx);
			atomicAdd(&_right_hand_y[target_node], dy);
			//right_hand[0] = param * width * 35;
		}
		else {/*move along y-axis costing less*/
			dx = param * abs(x_dis);
			dy = f_constraints[4] * height;
			/*keep the relative direction roughly*/
			if (dx*x_dis + dy*y_dis < 0) {
				dx *= -1;
				dy *= -1;
			}
			atomicAdd(&_right_hand_x[source_node], -dx);
			atomicAdd(&_right_hand_y[source_node], -dy);
			atomicAdd(&_right_hand_x[target_node], dx);
			atomicAdd(&_right_hand_y[target_node], dy);
			//right_hand[1] = height * 20;
		}
	}
	else {
		atomicAdd(&_right_hand_x[source_node], param*x_dis / f_constraints[5]);//* x_dis
		atomicAdd(&_right_hand_y[source_node], param*y_dis / f_constraints[5]);// * y_dis
		atomicAdd(&_right_hand_x[target_node], -param*x_dis / f_constraints[5]);//* x_dis
		atomicAdd(&_right_hand_y[target_node], -param*y_dis / f_constraints[5]);// *y_dis
	}

	return 5;
}
/*For Symmetry Constraints*/
__forceinline__ __device__ int SymmetryKernel(
	void* __restrict__ constraints,
	float* __restrict__ P_Opt_x,
	float* __restrict__ P_Opt_y,
	float* __restrict__ _right_hand_x,
	float* __restrict__ _right_hand_y) {

	float *f_constraints = (float *)constraints;
	int   *i_constraints = (int   *)constraints;
	/*node number in the relative shape = number of pairs of closest nodes*/
	int pid_size = i_constraints[2];
	int model_node_begin = 5;
	int other_node_begin = model_node_begin + pid_size;
	int r_length_begin = other_node_begin + pid_size, r_length_size = i_constraints[3];
	float param = f_constraints[4];
	float rotate_angle = f_constraints[0];
	float move_dis = f_constraints[1];

	int index = 0;
	for (int i = 0; i < pid_size; i++) {
		for (int j = i + 1; j < pid_size; j++) {
			/*direction of model edge(model node i to model node j)*/
			float dire_x = P_Opt_x[i_constraints[model_node_begin + i]] - P_Opt_x[i_constraints[model_node_begin + j]];
			float dire_y = P_Opt_y[i_constraints[model_node_begin + i]] - P_Opt_y[i_constraints[model_node_begin + j]];

			float eucli_dist_model = sqrtf(dire_x*dire_x + dire_y*dire_y);
			if (isnan(eucli_dist_model)) eucli_dist_model = 1;
			_rotate(rotate_angle, dire_x, dire_y);
			moveCenterToZero(move_dis, dire_x, dire_y);
			dire_x *= -1;
			//dire_x = dire_x / eucli_dist_model;
			//dire_y = dire_y / eucli_dist_model;

			_rotate(-rotate_angle, dire_x, dire_y);
			moveCenterToZero(-move_dis, dire_x, dire_y);

			float dx = param * dire_x / f_constraints[r_length_begin + index];
			float dy = param * dire_y / f_constraints[r_length_begin + index];

			//source
			atomicAdd(&_right_hand_x[i_constraints[other_node_begin + i]], dx);
			atomicAdd(&_right_hand_y[i_constraints[other_node_begin + i]], dy);
			//target
			atomicAdd(&_right_hand_x[i_constraints[other_node_begin + j]], -dx);
			atomicAdd(&_right_hand_y[i_constraints[other_node_begin + j]], -dy);

			index++;
		}
	}

	return 5 + 2 * i_constraints[2] + i_constraints[3];
}
/*For Atom Constraints*/
__forceinline__ __device__ int AtomKernel(
	void * __restrict__ constraints,
	float * __restrict__ P_Opt_x,
	float  * __restrict__ P_Opt_y,
	float  * __restrict__ _right_hand_x,
	float   * __restrict__ _right_hand_y) {//
	float *f_constraints = (float *)constraints;
	int   *i_constraints = (int   *)constraints;
	int warpid = threadIdx.x >> 5;
	int laneid = threadIdx.x - warpid << 5;

	//sum in every threads
	__shared__ int voting[4];
	if (laneid == 0)
		voting[warpid] = i_constraints[2];
	__syncthreads();
	if (voting[warpid] != i_constraints[2])
		voting[warpid] = -1;
	__syncthreads();

	//if this constraint does not assign te direction for this edge
	if (f_constraints[0] == 0 && f_constraints[1] == 0) {

		//original euclidean edge vector
		float dx = 0, dy = 0;
		dx = P_Opt_x[i_constraints[2]] - P_Opt_x[i_constraints[3]];//2 3
		dy = P_Opt_y[i_constraints[2]] - P_Opt_y[i_constraints[3]];
		if (dx == 0 && dy == 0) {
			dx = 0.0001f;
			dy = 0.0001f;
		}
		//weight para / normalization
		float para = f_constraints[4] / sqrtf(dx*dx + dy*dy);//4

															 //normalize and weight the edge vector
		dx *= para;
		dy *= para;

		float sum = dx, sum2 = dy;
		if (voting[warpid] > 0)
		{
			for (int i = 16; i >= 1; i >>= 1)
			{
				sum += __shfl_down(sum, i);//left shift variables in threads
				sum2 += __shfl_down(sum2, i);
			}
			if (laneid == 0)
			{
				atomicAdd(&_right_hand_x[i_constraints[2]], sum);
				atomicAdd(&_right_hand_y[i_constraints[2]], sum2);
			}
		}
		else {
			//edge's source
			atomicAdd(&_right_hand_x[i_constraints[2]], dx);
			atomicAdd(&_right_hand_y[i_constraints[2]], dy);
		}
		//edge's target
		atomicAdd(&_right_hand_x[i_constraints[3]], -dx);
		atomicAdd(&_right_hand_y[i_constraints[3]], -dy);
	}
	//this constraint assigns the direction to this edge
	//(f_constraints[0],f_constraints[1]) is the unit direction vector
	//f_constraints[4] is the weight para
	else {
		atomicAdd(&_right_hand_x[i_constraints[2]], 3.0f*f_constraints[4] * f_constraints[0]);
		atomicAdd(&_right_hand_y[i_constraints[2]], 3.0f*f_constraints[4] * f_constraints[1]);
		atomicAdd(&_right_hand_x[i_constraints[3]], -3.0f*f_constraints[4] * f_constraints[0]);
		atomicAdd(&_right_hand_y[i_constraints[3]], -3.0f*f_constraints[4] * f_constraints[1]);
	}

	return 3;
}

__forceinline__ __device__ int StressKernel(
	void * __restrict__ constraints,
	float* __restrict__ P_Opt_x,
	float* __restrict__ P_Opt_y,
	float* __restrict__ _right_hand_x,
	float* __restrict__ _right_hand_y) { }
__device__ Device_Stub stubs[] = { CrossingRemovalKernel, StressKernel, CircleKernel, EqualAngleKernel, NoverlapKernel, SymmetryKernel, AtomKernel }; ///

__global__ void LocalSteps(cuDWORD * __restrict__ constraints, float* __restrict__ P_Opt_x, float* __restrict__ P_Opt_y, float* __restrict__ _right_hand_x, float* __restrict__ _right_hand_y,
	int *constraints_offset, int const_num) {//float *right_hand
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (; i<const_num; i += gridDim.x * blockDim.x)
	{
		cuDWORD* i_constraints = constraints + constraints_offset[i];
		stubs[*i_constraints](i_constraints + 2, P_Opt_x, P_Opt_y, _right_hand_x, _right_hand_y);//i**(constraints + 1) *(constraints + 2)
	}
}

/*For Stress Constraints*/
__global__ void local_stress(
	float* __restrict__ P_Opt_x, float* __restrict__ P_Opt_y,
	float* __restrict__ _right_hand_x, float* __restrict__ _right_hand_y,

	float *__restrict__ dist, int n
) {
	unsigned stepping = blockDim.x;
	unsigned j = blockIdx.x;
	static __shared__ float shared[64];

	float sum = 0, sum2 = 0;
	unsigned wid = threadIdx.x >> 5;	// warp ID
	unsigned lane = threadIdx.x - (wid << 5);
#pragma unroll
	for (; j < n; j += gridDim.x) {
		sum = 0; sum2 = 0;
#pragma unroll
		for (unsigned i = threadIdx.x; i < n; i += stepping)
		{
			if (i != j) {

				float t = rsqrt(pow(_right_hand_x[j] - _right_hand_x[i], 2.f) + pow(_right_hand_y[j] - _right_hand_y[i], 2.f));

				if (isinf(t) || isnan(t)) t = 1;
				sum += ((t / dist(j, i))) * (_right_hand_x[j] - _right_hand_x[i]);
				sum2 += ((t / dist(j, i))) * (_right_hand_y[j] - _right_hand_y[i]);
			}
		}

#pragma unroll
		for (unsigned offset = 16; offset > 0; offset >>= 1)
		{
			sum += __shfl_down(sum, offset);
			sum2 += __shfl_down(sum2, offset);
		}
		if (lane == 0)
		{
			shared[wid + wid] = sum;
			shared[wid + wid + 1] = sum2;
		}
		__syncthreads();

		sum = (threadIdx.x < 32) ? shared[lane + lane] : 0;// group to warp 1;
		sum2 = (threadIdx.x < 32) ? shared[lane + lane + 1] : 0;// group to warp 1;

		if (wid == 0) {
#pragma unroll
			for (unsigned offset = 16; offset > 0; offset >>= 1)
			{
				sum += __shfl_down(sum, offset);
				sum2 += __shfl_down(sum2, offset);
			}
		}
		if (threadIdx.x == 0)
		{
			P_Opt_x[j] = sum;
			P_Opt_y[j] = sum2;
		}
	}
	sum = 0; sum2 = 0;
}
__global__ void cuMemchecker(float *mem, float *mem2) {
	mem[0] = 0;
}
__global__ void jacobiPreconditioner(float *M, float *diag, int n, int innersize) {
	//	int i = threadIdx.x;
	int j = blockIdx.x;
	for (; j < n; j += gridDim.x) {
		for (int i = threadIdx.x; i < n; i += blockDim.x) {
			if (i != j) {
				M[j*innersize + i] /= M[j*innersize + j];
			}
		}
		__syncthreads();
		if (threadIdx.x == 0)
		{
			diag[j] = M[j*innersize + j];
			M[j*innersize + j] = 1;
		}
	}
}
__global__ void jacobiPrecondImpl(float *diag, float *x, float *y, int n) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n) {
		x[i] /= diag[i];
		y[i] /= diag[i];

	}
	else if (i<2 * n) {

	}

}
DWORD WINAPI finalize(LPVOID);


class CudaCore {

private:
	viennacl::matrix<float> system_matrix;
	viennacl::vector<float> right_hand, right_hand_x, right_hand_y;
	viennacl::vector<float> result, result_x, result_y;
	float *d_right_x;
	float *d_right_y;
	float* res_x;
	float* res_y;
	bool runonce = true;
	viennacl::linalg::cg_solver<viennacl::vector<float>> *solver;
	void *d_mat;
	hipStream_t nonBlocking;

	cuDWORD *d_constraints;

	int *d_const_idx;
	float *d_dist = 0;
	float *diag;
	//J
	float *d_J_val;
	int *d_J_RowPtr, *d_J_ColIdx;
	int n_data = -1, constraints_size, constraints_length, val_size;
	int const_num;

	int blocks, threads = 128;

public:
	cuDWORD *d_gc1;
	int *d_gc2;
	int MaxIter = 100;

	CudaCore()
	{}

	//
	void init(float* sysm, int n_data, cuDWORD* constraints, int* const_idx, int constraint_num, float **dists)
	{

		if (d_dist == 0 || this->n_data != n_data) {
			hipStreamCreateWithFlags(&nonBlocking, hipStreamNonBlocking);

			this->n_data = n_data; // n_data is the rows of L, = 2 * nodenum 

			system_matrix = viennacl::matrix<float>(n_data, n_data, viennacl::context(viennacl::CUDA_MEMORY));
			d_mat = system_matrix.handle().cuda_handle().get();
			hipMalloc(&diag, n_data * sizeof(float));
			hipMalloc(&d_dist, sizeof(float) * (n_data*(n_data - 1)) / 2);
			float *d_dist_ptr = d_dist;
			for (int i = 0; i < n_data; i++) {
				int to_cpy = n_data - i - 1;
				hipMemcpyAsync(d_dist_ptr, dists[i] + i + 1, sizeof(float) * to_cpy, hipMemcpyHostToDevice, nonBlocking);
				d_dist_ptr += to_cpy;
			}

			solver = new viennacl::linalg::cg_solver<viennacl::vector<float>>(viennacl::linalg::cg_tag(1e-5, 15));//, 80 for 12000

			right_hand_x = viennacl::vector<float>(n_data, viennacl::context(viennacl::CUDA_MEMORY));//d_right
			right_hand_y = viennacl::vector<float>(n_data, viennacl::context(viennacl::CUDA_MEMORY));//d_right

			d_right_x = (float *)right_hand_x.handle().cuda_handle().get();
			d_right_y = (float *)right_hand_y.handle().cuda_handle().get();

			result_x = viennacl::vector<float>(n_data, viennacl::context(viennacl::CUDA_MEMORY));//d_right;
			result_y = viennacl::vector<float>(n_data, viennacl::context(viennacl::CUDA_MEMORY));//d_right;

			res_x = (float*)result_x.handle().cuda_handle().get();
			res_y = (float*)result_y.handle().cuda_handle().get();
		}
		const_num = constraint_num;
		constraints_length = const_idx[const_num];
		hipMalloc(&d_constraints, constraints_length * sizeof(cuDWORD));
		hipMalloc(&d_const_idx, const_num * sizeof(int));
		hipMemcpy(d_constraints, constraints, constraints_length * sizeof(cuDWORD), hipMemcpyDefault);
		//	delete[] constraints;
		hipMemcpy(d_const_idx, const_idx, sizeof(int)*const_num, hipMemcpyDefault);
		delete[] const_idx;


		int internal_size = system_matrix.internal_size1();
		for (int i = 0; i < n_data; i++) {
			hipMemcpyAsync(
				((cuDWORD *)d_mat) + i *internal_size,
				sysm + n_data * i, n_data * sizeof(float),
				hipMemcpyHostToDevice// , nonBlocking
			);
		}
		//hipStreamSynchronize(nonBlocking);
		//	jacobiPreconditioner<<<128,128>>>((float*)d_mat, diag, n_data, internal_size);
	}
	int t = 0;


	void Solve(float *data_x, float *data_y,
		float *out_x, float *out_y, int iters, int* es, float* shortest_path) {
		if (runonce) {
			hipMemcpy(res_x, out_x, sizeof(float)*n_data, hipMemcpyHostToDevice);
			hipMemcpy(res_y, out_y, sizeof(float)*n_data, hipMemcpyHostToDevice);
			runonce = false;
		}
		std::cout << "iterations: " << iters << std::endl;
		for (int i = 0; i < iters; i++) {
			if (i == 2)
				solver->tag_ = viennacl::linalg::cg_tag(1e-5, 13);
			else if (i == 10)//18-19
				solver->tag_ = viennacl::linalg::cg_tag(1e-5, 10);
			else if (i == 15)
				solver->tag_ = viennacl::linalg::cg_tag(1e-5, 8);

			hipMemset(d_right_x, 0, sizeof(float)*(n_data));
			hipMemset(d_right_y, 0, sizeof(float)*(n_data));
			local_stress << <128, 1024 >> >(d_right_x, d_right_y, res_x, res_y, d_dist, n_data);
			LocalSteps << <128, 128 >> >(d_constraints, res_x, res_y, d_right_x, d_right_y, d_const_idx, const_num); //

			(*solver)(system_matrix, right_hand_x, right_hand_y);

			hipMemset(res_x, 0, sizeof(float)*(n_data));
			hipMemset(res_y, 0, sizeof(float)*(n_data));
			local_stress << <128, 1024 >> >(res_x, res_y, d_right_x, d_right_y, d_dist, n_data);
			LocalSteps << <128, 128 >> >(d_constraints, d_right_x, d_right_y, res_x, res_y, d_const_idx, const_num); //

			(*solver).operator()<decltype(system_matrix), true>(system_matrix, result_x, result_y);


		}

		hipMemcpyAsync(
			out_x,
			res_x,
			n_data * sizeof(float),
			hipMemcpyDeviceToHost
		);
		hipMemcpyAsync(
			out_y,
			res_y,
			n_data * sizeof(float),
			hipMemcpyDeviceToHost
		);
		/*
		d_gc1 = d_constraints;
		d_gc2 = d_const_idx;

		CreateThread(0, 0, finalize, this, 0, 0);
		*//*d_constraints = 0;
		d_const_idx = 0;*/

		hipFree(d_constraints);

		d_constraints = 0;

		hipFree(d_const_idx);

		d_const_idx = 0;
	}


	~CudaCore() {
		if (!right_hand_x.empty())
		{

			if (!d_constraints)

				hipFree(d_constraints);

			d_constraints = 0;

			if (!d_const_idx)

				hipFree(d_const_idx);

			d_const_idx = 0;

			if (!d_dist)

				hipFree(d_dist);

			d_dist = 0;

			delete solver;
		}
	}
};
DWORD WINAPI finalize(LPVOID param) {
	CudaCore* core = ((CudaCore*)param);
	hipFree(core->d_gc1);
	hipFree(core->d_gc2);
	return 0;
}
CudaSolver::CudaSolver() {
	core = new CudaCore();

}
CudaSolver::~CudaSolver() {
	delete core;
}

void CudaSolver::init(float *mat, int n_mat, cuDWORD* constraints, int* const_idx, int constraint_size, float **dists) {
	core->init(mat, n_mat, constraints, const_idx, constraint_size, dists);
}

void CudaSolver::Solve(float *right_hand_x, float *right_hand_y, float* P_Out_x, float* P_Out_y, int iters,
	int* es, float* shortest_path) {
	core->Solve(right_hand_x, right_hand_y, P_Out_x, P_Out_y, iters, es, shortest_path);
}



void CudaSolver::setMaxIter(int mi) {
	core->MaxIter = mi;
}
__inline__ __device__  void _rotate(float alpha, float &edge_X, float &edge_Y) {
	float thow = (sqrtf(edge_X * edge_X + edge_Y * edge_Y));
	float p_angle = 0; // x==0, y==0
	if (edge_X > 0) {
		p_angle = atanf(edge_Y / edge_X);
	}
	else if (edge_X < 0 && edge_Y >= 0) {
		p_angle = atanf(edge_Y / edge_X) + 3.14159;
	}
	else if (edge_X< 0 && edge_Y < 0) {
		p_angle = atanf(edge_Y / edge_X) - 3.14159;
	}
	else if (edge_X == 0 && edge_Y < 0) {
		p_angle = -3.14159 / 2;
	}
	else if (edge_X == 0 && edge_Y > 0) {
		p_angle = 3.14159 / 2;
	}
	edge_X = thow * cos(p_angle - alpha);
	edge_Y = thow * sin(p_angle - alpha);
}
__inline__ __device__  void moveCenterToZero(float center_x, float &edge_X, float &edge_Y) {
	edge_X -= center_x;
}
__inline__ __device__ void ShapeMatching(float *relative_shape,
	float *P_Opt_x, float *P_Opt_y, int *pid, int node_num) {

	float X0cmx = 0.0f, X0cmy = 0.0f, Xcmx = 0.0f, Xcmy = 0.0f;
	for (int i = 0; i < node_num; i++) {
		X0cmx += relative_shape[2 * i + 0];
		X0cmy += relative_shape[2 * i + 1];

		Xcmx += P_Opt_x[pid[i]];
		Xcmy += P_Opt_y[pid[i]];
	}
	X0cmx /= node_num;
	X0cmy /= node_num;
	Xcmx /= node_num;
	Xcmy /= node_num;

	float Apqa = 0.0f, Apqb = 0.0f, Apqc = 0.0f, Apqd = 0.0f;
	for (int i = 0; i < node_num; i++) {
		float qix = relative_shape[2 * i + 0] - X0cmx;
		float qiy = relative_shape[2 * i + 1] - X0cmx;
		float pix = P_Opt_x[pid[i]] - Xcmx;
		float piy = P_Opt_y[pid[i]] - Xcmy;

		Apqa += pix*qix;
		Apqb += pix*qiy;
		Apqc += piy*qix;
		Apqd += piy*qiy;
	}

	float Sa, Sb, Sc, Sd;
	Sa = Apqa*Apqa + Apqc*Apqc;
	Sb = Apqa*Apqb + Apqc*Apqd;
	Sc = Apqa*Apqb + Apqc*Apqd;
	Sd = Apqb*Apqb + Apqd*Apqd;

	//matrix sqrt
	float tao_real = Sa + Sd, tao_i = 0;
	float sigma_real = Sa*Sd - Sb*Sc, sigma_i = 0;
	float s_real = 0, s_i = 0;
	if (sigma_real >= 0) {
		s_real = sqrt(sigma_real);
	}
	else {
		s_i = sqrt(abs(sigma_real));
	}

	float t_real = tao_real + 2 * s_real, t_i = tao_i + 2 * s_i;
	if (t_real >= 0) {
		t_real = sqrt(t_real);
	}
	else {
		t_i = sqrt(abs(t_real));
	}

	float _t_real = t_real / (t_real*t_real - t_i*t_i), _t_i = -t_i / (t_real*t_real - t_i*t_i);
	Sa = _t_real*(Sa + s_real) - _t_i*s_i;
	Sb = _t_real*(Sb);
	Sc = _t_real*(Sc);
	Sd = _t_real*(Sd + s_real) - _t_i*s_i;



	float ISa, ISb, ISc, ISd;
	float S_trace = Sa*Sd - Sb*Sc;
	if (S_trace == 0.0f) {
		S_trace = 0.0001;
	}
	ISa = Sd / S_trace;
	ISb = -Sb / S_trace;
	ISc = -Sc / S_trace;
	ISd = Sa / S_trace;

	float Ra, Rb, Rc, Rd;
	Ra = Apqa*ISa + Apqb*ISc;
	Rb = Apqa*ISb + Apqb*ISd;
	Rc = Apqc*ISa + Apqd*ISc;
	Rd = Apqc*ISb + Apqd*ISd;

	for (int i = 0; i < node_num; i++) {
		float Xi0x = relative_shape[2 * i + 0];
		float Xi0y = relative_shape[2 * i + 1];

		float gix = Ra*(Xi0x - X0cmx) + Rb*(Xi0y - X0cmy) + Xcmx;
		float giy = Rc*(Xi0x - X0cmx) + Rd*(Xi0y - X0cmy) + Xcmy;

		relative_shape[2 * i + 0] = gix;
		relative_shape[2 * i + 1] = giy;
	}
}
